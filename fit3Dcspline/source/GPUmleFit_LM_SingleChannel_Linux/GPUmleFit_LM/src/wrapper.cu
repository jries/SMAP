//file GPUmleFit_LM_sCMOS.cu
//author Yiming Li
//date 20170301

 //brief Wrap the Cuda kernel calls as standard external C functions.  This allows the kernels to be
 // called without doing anything special in the C code and simplifies building the code.
 //

//Terms of Use 
//
//This file is part of GPUmleFit_LM. 
//
//GPUmleFit_LM Fitter is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version. 
//
//GPUmleFit_LM Fitter is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for more details. 
//
//You should have received a copy of the GNU General Public License along with GPUmleFit_LM Fitter. If not, see <http://www.gnu.org/licenses/>. 
//
//Additional permission under GNU GPL version 3 section 7 

#include <definitions.h>
#include "GPUmleFit_LM_EMCCD.h"
#include "GPUgaussLib.h"
#include "GPUmleFit_LM_sCMOS.h"

//EMCCD wrapper

//*******************************************************************************************
extern void kernel_MLEFit_EMCCD_wrapper(dim3 dimGrid, dim3 dimBlock, const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits) 
{
/*!
 *  \brief Basic maximum likelihood estimator fit based kernel
 *  \param dimGrid number of blocks 
 *  \param dimBlock number of threads per block
 *  \param d_data an array of subregions to be processed copied into video memory
 *  \param PSFSigma the sigma value to use for the point spread function
 *  \param sz nxn size of the subregion
 *  \param iterations maximum allowed iterations before aborting fitting
 *  \param d_Parameters pointer to result array of fitted parameters, x, y coords, etc.
 *  \param d_CRLBs pointer to result array of Cramer-Rao lower bound estimates 
 *  \param d_LogLikelihood pointer to result array of loglikelihood estimates of fitting
 *  \param Nfits number of subregions to fit
 */

	kernel_MLEFit_LM_EMCCD<<<dimGrid, dimBlock>>>(d_data, PSFSigma, sz, iterations, d_Parameters, d_CRLBs, d_LogLikelihood, Nfits);
	
}

//*******************************************************************************************
extern void kernel_MLEFit_sigma_EMCCD_wrapper(dim3 dimGrid, dim3 dimBlock, const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits) 
{
/*!
 *  \brief Basic maximum likelihood estimator fit based kernel
 *  \param dimGrid number of blocks 
 *  \param dimBlock number of threads per block
 *  \param d_data an array of subregions to be processed copied into video memory
 *  \param PSFSigma the sigma value to use for the point spread function
 *  \param sz nxn size of the subregion
 *  \param iterations maximum allowed iterations before aborting fitting
 *  \param d_Parameters pointer to result array of fitted parameters, x, y coords, etc.
 *  \param d_CRLBs pointer to result array of Cramer-Rao lower bound estimates 
 *  \param d_LogLikelihood pointer to result array of loglikelihood estimates of fitting
 *  \param Nfits number of subregions to fit
 */
	kernel_MLEFit_LM_Sigma_EMCCD<<<dimGrid, dimBlock>>>(d_data, PSFSigma, sz, iterations, d_Parameters, d_CRLBs, d_LogLikelihood, Nfits);
	
}

//*******************************************************************************************
extern void kernel_MLEFit_z_EMCCD_wrapper(dim3 dimGrid, dim3 dimBlock, const float *d_data, const float PSFSigma_x, const float Ax, const float Ay, const float Bx, 
		const float By, const float gamma, const float d, const float PSFSigma_y, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits) 
{
/*!
 *  \brief Basic maximum likelihood estimator fit based kernel
 *  \param dimGrid number of blocks 
 *  \param dimBlock number of threads per block
 *  \param d_data an array of subregions to be processed copied into video memory
 *  \param PSFSigma_x the sigma value to use for the point spread function on the x axis
 *  \param Ax ???
 *  \param Ay ???
 *  \param Bx ???
 *  \param By ???
 *  \param gamma ???
 *  \param d ???
 *  \param PSFSigma_y the sigma value to use for the point spread function on the y axis
 *  \param sz nxn size of the subregion
 *  \param iterations maximum allowed iterations before aborting fitting
 *  \param d_Parameters pointer to result array of fitted parameters, x, y coords, etc.
 *  \param d_CRLBs pointer to result array of Cramer-Rao lower bound estimates 
 *  \param d_LogLikelihood pointer to result array of loglikelihood estimates of fitting
 *  \param Nfits number of subregions to fit
 */
	kernel_MLEFit_LM_z_EMCCD<<<dimGrid, dimBlock>>>(d_data, PSFSigma_x, Ax, Ay, Bx, By, gamma, d, PSFSigma_y, sz, iterations, d_Parameters, d_CRLBs, d_LogLikelihood, Nfits);
}

//*******************************************************************************************
extern void kernel_MLEFit_sigmaxy_EMCCD_wrapper(dim3 dimGrid, dim3 dimBlock, const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits) 
{
	/*!
 *  \brief Basic maximum likelihood estimator fit based kernel
 *  \param dimGrid number of blocks 
 *  \param dimBlock number of threads per block
 *  \param d_data an array of subregions to be processed copied into video memory
 *  \param PSFSigma the sigma value to use for the point spread function
 *  \param sz nxn size of the subregion
 *  \param iterations maximum allowed iterations before aborting fitting
 *  \param d_Parameters pointer to result array of fitted parameters, x, y coords, etc.
 *  \param d_CRLBs pointer to result array of Cramer-Rao lower bound estimates 
 *  \param d_LogLikelihood pointer to result array of loglikelihood estimates of fitting
 *  \param Nfits number of subregions to fit
 */
	kernel_MLEFit_LM_sigmaxy_EMCCD<<<dimGrid, dimBlock>>>(d_data, PSFSigma, sz, iterations, d_Parameters, d_CRLBs, d_LogLikelihood, Nfits);
}


extern void kernel_splineMLEFit_z_EMCCD_wrapper(dim3 dimGrid, dim3 dimBlock, const float *d_data,const float *d_coeff, const int spline_xsize, const int spline_ysize, const int spline_zsize, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,float initZ, const int Nfits) 
/*!
 *  \brief Basic maximum likelihood estimator fit based kernel
 *  \param dimGrid number of blocks 
 *  \param dimBlock number of threads per block
 *  \param d_data an array of subregions to be processed copied into video memory
 *  \param d_coeff spline coefficient
 *  \param sz nxn size of the subregion
 *  \param iterations maximum allowed iterations before aborting fitting
 *  \param d_Parameters pointer to result array of fitted parameters, x, y coords, etc.
 *  \param d_CRLBs pointer to result array of Cramer-Rao lower bound estimates 
 *  \param d_LogLikelihood pointer to result array of loglikelihood estimates of fitting
 *  \param Nfits number of subregions to fit
 */
{
	kernel_splineMLEFit_z_EMCCD<<<dimGrid, dimBlock>>>(d_data, d_coeff, spline_xsize, spline_ysize, spline_zsize, sz, iterations, d_Parameters, d_CRLBs, d_LogLikelihood,initZ, Nfits);
}


//sCMOS wrapper
//*******************************************************************************************
extern void kernel_MLEFit_sCMOS_wrapper(dim3 dimGrid, dim3 dimBlock, const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits, const float *d_varim) 
{
/*!
 *  \brief Basic maximum likelihood estimator fit based kernel
 *  \param dimGrid number of blocks 
 *  \param dimBlock number of threads per block
 *  \param d_data an array of subregions to be processed copied into video memory
 *  \param PSFSigma the sigma value to use for the point spread function
 *  \param sz nxn size of the subregion
 *  \param iterations maximum allowed iterations before aborting fitting
 *  \param d_Parameters pointer to result array of fitted parameters, x, y coords, etc.
 *  \param d_CRLBs pointer to result array of Cramer-Rao lower bound estimates 
 *  \param d_LogLikelihood pointer to result array of loglikelihood estimates of fitting
 *  \param Nfits number of subregions to fit
 *  \param d_varim variance map for sCMOS
 */

	kernel_MLEFit_LM_sCMOS<<<dimGrid, dimBlock>>>(d_data, PSFSigma, sz, iterations, d_Parameters, d_CRLBs, d_LogLikelihood, Nfits,d_varim);
}

//*******************************************************************************************
extern void kernel_MLEFit_sigma_sCMOS_wrapper(dim3 dimGrid, dim3 dimBlock, const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits, const float *d_varim) 
{
/*!
 *  \brief Basic maximum likelihood estimator fit based kernel
 *  \param dimGrid number of blocks 
 *  \param dimBlock number of threads per block
 *  \param d_data an array of subregions to be processed copied into video memory
 *  \param PSFSigma the sigma value to use for the point spread function
 *  \param sz nxn size of the subregion
 *  \param iterations maximum allowed iterations before aborting fitting
 *  \param d_Parameters pointer to result array of fitted parameters, x, y coords, etc.
 *  \param d_CRLBs pointer to result array of Cramer-Rao lower bound estimates 
 *  \param d_LogLikelihood pointer to result array of loglikelihood estimates of fitting
 *  \param Nfits number of subregions to fit
 *  \param d_varim variance map for sCMOS
 */
	kernel_MLEFit_LM_Sigma_sCMOS<<<dimGrid, dimBlock>>>(d_data, PSFSigma, sz, iterations, d_Parameters, d_CRLBs, d_LogLikelihood, Nfits,d_varim);
}

//*******************************************************************************************
extern void kernel_MLEFit_z_sCMOS_wrapper(dim3 dimGrid, dim3 dimBlock, const float *d_data, const float PSFSigma_x, const float Ax, const float Ay, const float Bx, 
		const float By, const float gamma, const float d, const float PSFSigma_y, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits, const float *d_varim) 
{
/*!
 *  \brief Basic maximum likelihood estimator fit based kernel
 *  \param dimGrid number of blocks 
 *  \param dimBlock number of threads per block
 *  \param d_data an array of subregions to be processed copied into video memory
 *  \param PSFSigma_x the sigma value to use for the point spread function on the x axis
 *  \param Ax ???
 *  \param Ay ???
 *  \param Bx ???
 *  \param By ???
 *  \param gamma ???
 *  \param d ???
 *  \param PSFSigma_y the sigma value to use for the point spread function on the y axis
 *  \param sz nxn size of the subregion
 *  \param iterations maximum allowed iterations before aborting fitting
 *  \param d_Parameters pointer to result array of fitted parameters, x, y coords, etc.
 *  \param d_CRLBs pointer to result array of Cramer-Rao lower bound estimates 
 *  \param d_LogLikelihood pointer to result array of loglikelihood estimates of fitting
 *  \param Nfits number of subregions to fit
 *  \param d_varim variance map for sCMOS
 */
	kernel_MLEFit_LM_z_sCMOS<<<dimGrid, dimBlock>>>(d_data, PSFSigma_x, Ax, Ay, Bx, By, gamma, d, PSFSigma_y, sz, iterations, d_Parameters, d_CRLBs, d_LogLikelihood, Nfits,d_varim);
}

//*******************************************************************************************
extern void kernel_MLEFit_sigmaxy_sCMOS_wrapper(dim3 dimGrid, dim3 dimBlock, const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits, const float *d_varim) 
{
	/*!
 *  \brief Basic maximum likelihood estimator fit based kernel
 *  \param dimGrid number of blocks 
 *  \param dimBlock number of threads per block
 *  \param d_data an array of subregions to be processed copied into video memory
 *  \param PSFSigma the sigma value to use for the point spread function
 *  \param sz nxn size of the subregion
 *  \param iterations maximum allowed iterations before aborting fitting
 *  \param d_Parameters pointer to result array of fitted parameters, x, y coords, etc.
 *  \param d_CRLBs pointer to result array of Cramer-Rao lower bound estimates 
 *  \param d_LogLikelihood pointer to result array of loglikelihood estimates of fitting
 *  \param Nfits number of subregions to fit
 *  \param d_varim variance map for sCMOS
 */
	kernel_MLEFit_LM_sigmaxy_sCMOS<<<dimGrid, dimBlock>>>(d_data, PSFSigma, sz, iterations, d_Parameters, d_CRLBs, d_LogLikelihood, Nfits,d_varim);
}


extern void kernel_splineMLEFit_z_sCMOS_wrapper(dim3 dimGrid, dim3 dimBlock, const float *d_data,const float *d_coeff, const int spline_xsize, const int spline_ysize, const int spline_zsize, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,float initZ, const int Nfits, const float *d_varim) 
{
	/*!
 *  \brief Basic maximum likelihood estimator fit based kernel
 *  \param dimGrid number of blocks 
 *  \param dimBlock number of threads per block
 *  \param d_data an array of subregions to be processed copied into video memory
 *  \param d_coeff spline coefficient
 *  \param sz nxn size of the subregion
 *  \param iterations maximum allowed iterations before aborting fitting
 *  \param d_Parameters pointer to result array of fitted parameters, x, y coords, etc.
 *  \param d_CRLBs pointer to result array of Cramer-Rao lower bound estimates 
 *  \param d_LogLikelihood pointer to result array of loglikelihood estimates of fitting
 *  \param Nfits number of subregions to fit
 *  \param d_varim variance map for sCMOS
 */
	kernel_splineMLEFit_z_sCMOS<<<dimGrid, dimBlock>>>(d_data, d_coeff, spline_xsize, spline_ysize, spline_zsize, sz, iterations, d_Parameters, d_CRLBs, d_LogLikelihood,initZ, Nfits, d_varim);
}
