#include "hip/hip_runtime.h"
//file GPUmleFit_LM_sCMOS.cu
//author Yiming Li
//date 20170301
//brief This is GPUmleFit_LM for Gaussian model and Spline model.

//Terms of Use 
//
//This file is part of GPUmleFit_LM. 
//
//GPUmleFit_LM Fitter is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version. 
//
//GPUmleFit_LM Fitter is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for more details. 
//
//You should have received a copy of the GNU General Public License along with GPUmleFit_LM Fitter. If not, see <http://www.gnu.org/licenses/>. 
//
//Additional permission under GNU GPL version 3 section 7 

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "mex.h"
#include "hip/hip_runtime.h"
#include "definitions.h"
#include "MatInvLib.h"

#include "GPUsplineLib.cuh"
#include "GPUgaussLib.cuh"
#include "GPUmleFit_LM_sCMOS.h"

__global__ void kernel_MLEFit_LM_sCMOS(const float *d_data,const float PSFSigma, const int sz, const int iterations, 
	float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits, const float *d_varim){
		/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param PSFSigma sigma of the point spread function
	 * \param sz nxn size of the subregion to fit
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 * \d_varim variance map of scmos
	 */

		const int NV=NV_P;
		float M[NV*NV],Diag[NV], Minv[NV*NV];
		int tx = threadIdx.x;
		int bx = blockIdx.x;
		int BlockSize = blockDim.x;
		int ii, jj, kk, ll, l, m, i;


		float model, data;
		float Div;

		float newTheta[NV],oldTheta[NV];
		float newLambda = INIT_LAMBDA, oldLambda = INIT_LAMBDA, mu;
	    float newUpdate[NV] = {1e13, 1e13, 1e13, 1e13},oldUpdate[NV] = {1e13, 1e13, 1e13, 1e13};
		float maxJump[NV]={1.0,1.0,100,20};
		float newDudt[NV] ={0};

		float newErr = 1e13, oldErr = 1e13;

		float jacobian[NV]={0};
		float hessian[NV*NV]={0};
		float t1,t2;

		float Nmax;
		int errFlag=0;
		float L[NV*NV] = {0}, U[NV*NV] = {0};


		//Prevent read/write past end of array
		if ((bx*BlockSize+tx)>=Nfits) return;

		for (ii=0;ii<NV*NV;ii++)M[ii]=0;
		for (ii=0;ii<NV*NV;ii++)Minv[ii]=0;

		//copy in data
		const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
		const float *s_varim = d_varim+(sz*sz*bx*BlockSize+sz*sz*tx);

		//initial values
		kernel_CenterofMass2D(sz, s_data, &newTheta[0], &newTheta[1]);
		kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &newTheta[3]);
		newTheta[2]=max(0.0, (Nmax-newTheta[3])*2*pi*PSFSigma*PSFSigma);
		newTheta[3] = max(newTheta[3],0.01);

		maxJump[2]=max(newTheta[2],maxJump[2]);

		maxJump[3]=max(newTheta[3],maxJump[3]);

		for (ii=0;ii<NV;ii++)oldTheta[ii]=newTheta[ii];

		//updateFitValues
		newErr = 0;
		memset(jacobian,0,NV*sizeof(float));
		memset(hessian,0,NV*NV*sizeof(float));
		for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
			kernel_DerivativeGauss2D(ii,jj,PSFSigma,newTheta,newDudt,&model);
			model +=s_varim[sz*jj+ii];
			data=s_data[sz*jj+ii]+s_varim[sz*jj+ii];

			if (data>0)
				newErr = newErr + 2*((model-data)-data*log(model/data));
			else
			{
				newErr = newErr + 2*model;
				data = 0;
			}

			t1 = 1-data/model;
			for (l=0;l<NV;l++){
				jacobian[l]+=t1*newDudt[l];
			}

			t2 = data/pow(model,2);
			for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
				hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
				hessian[m*NV+l] = hessian[l*NV+m];
			}
		}

		for (kk=0;kk<iterations;kk++) {//main iterative loop

			if(abs((newErr-oldErr)/newErr)<TOLERANCE){
				//CONVERGED;
				break;
			}
			else{
				if(newErr>ACCEPTANCE*oldErr){
					//copy Fitdata

					for (i=0;i<NV;i++){
						newTheta[i]=oldTheta[i];
						newUpdate[i]=oldUpdate[i];
					}
					newLambda = oldLambda;
					newErr = oldErr;
					mu = max( (1 + newLambda*SCALE_UP)/(1 + newLambda),1.3f);         
					newLambda = SCALE_UP*newLambda;

				}
				else if(newErr<oldErr&&errFlag==0){
					newLambda = SCALE_DOWN*newLambda;
				    mu = 1+newLambda;
				}

				for (i=0;i<NV;i++){
					hessian[i*NV+i]=hessian[i*NV+i]*mu;
				}
				memset(L,0,NV*sizeof(float));
				memset(U,0,NV*sizeof(float));
				errFlag = kernel_cholesky(hessian,NV,L,U);
				if (errFlag ==0){

					for (i=0;i<NV;i++){
						oldTheta[i]=newTheta[i];
						oldUpdate[i] = newUpdate[i];
					}
					oldLambda = newLambda;
					oldErr=newErr;

					kernel_luEvaluate(L,U,jacobian,NV,newUpdate);	
					
					//updateFitParameters
					for (ll=0;ll<NV;ll++){
						if (newUpdate[ll]/oldUpdate[ll]< -0.5f){
							maxJump[ll] = maxJump[ll]*0.5;
						}
					    newUpdate[ll] = newUpdate[ll]/(1+fabs(newUpdate[ll]/maxJump[ll]));
						newTheta[ll] = newTheta[ll]-newUpdate[ll];
					}

					//restrict range
					newTheta[0] = max(newTheta[0],(float(sz)-1)/2-sz/4.0);
					newTheta[0] = min(newTheta[0],(float(sz)-1)/2+sz/4.0);
					newTheta[1] = max(newTheta[1],(float(sz)-1)/2-sz/4.0);
					newTheta[1] = min(newTheta[1],(float(sz)-1)/2+sz/4.0);
					newTheta[2] = max(newTheta[2],1.0);
					newTheta[3] = max(newTheta[3],0.01);
					


					newErr = 0;
					memset(jacobian,0,NV*sizeof(float));
					memset(hessian,0,NV*NV*sizeof(float));
					for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
						//calculating derivatives
						kernel_DerivativeGauss2D(ii,jj,PSFSigma,newTheta,newDudt,&model);
						model +=s_varim[sz*jj+ii];
						data=s_data[sz*jj+ii]+s_varim[sz*jj+ii];		

						if (data>0)
							newErr = newErr + 2*((model-data)-data*log(model/data));
						else 
						{
							newErr = newErr + 2*model;
							data = 0;
						}

						t1 = 1-data/model;
						for (l=0;l<NV;l++){
							jacobian[l]+=t1*newDudt[l];
						}

						t2 = data/pow(model,2);
						for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
							hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
							hessian[m*NV+l] = hessian[l*NV+m];
						}
					}
				}
				else
				{
					mu = max( (1 + newLambda*SCALE_UP)/(1 + newLambda),1.3f);         
					newLambda = SCALE_UP*newLambda;
				}
			}
		}
		//output iteration
		d_Parameters[Nfits*NV+BlockSize*bx+tx]=kk;
		// Calculating the CRLB and LogLikelihood
		Div=0.0f;
		for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
			kernel_DerivativeGauss2D(ii,jj,PSFSigma,newTheta,newDudt,&model);
			model +=s_varim[sz*jj+ii];
			data=s_data[sz*jj+ii]+s_varim[sz*jj+ii];	

			//Building the Fisher Information Matrix
			for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
				M[kk*NV+ll]+= newDudt[ll]*newDudt[kk]/model;
				M[ll*NV+kk]=M[kk*NV+ll];
			}

			//LogLikelyhood
			if (model>0)
				if (data>0)Div+=data*log(model)-model-data*log(data)+data;
				else
					Div+=-model;
		}

		// Matrix inverse (CRLB=F^-1) and output assigments
		kernel_MatInvN(M, Minv, Diag, NV);
		//write to global arrays
		for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=newTheta[kk];
		for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
		d_LogLikelihood[BlockSize*bx+tx] = Div;

		return;
}

//*********************************************************************************************************************************************

__global__ void kernel_MLEFit_LM_Sigma_sCMOS(const float *d_data,const float PSFSigma, const int sz, const int iterations, 
	float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits, const float *d_varim){
		/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param PSFSigma sigma of the point spread function
	 * \param sz nxn size of the subregion to fit
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 * \d_varim variance map of scmos
	 */

		const int NV=NV_PS;
		float M[NV*NV],Diag[NV], Minv[NV*NV];
		int tx = threadIdx.x;
		int bx = blockIdx.x;
		int BlockSize = blockDim.x;
		int ii, jj, kk, ll, l, m, i;

		float model, data;
		float Div;

		float newTheta[NV],oldTheta[NV];
		float newLambda = INIT_LAMBDA, oldLambda = INIT_LAMBDA, mu;
		float newUpdate[NV] = {1e13, 1e13, 1e13, 1e13, 1e13},oldUpdate[NV] = {1e13, 1e13, 1e13, 1e13, 1e13};
		float maxJump[NV]={1.0,1.0,100,20,0.5};
		float newDudt[NV] ={0};

		float newErr = 1e12, oldErr = 1e13;

		float jacobian[NV]={0};
		float hessian[NV*NV]={0};
		float t1,t2;

		float Nmax;
		int errFlag=0;
		float L[NV*NV] = {0}, U[NV*NV] = {0};


		//Prevent read/write past end of array
		if ((bx*BlockSize+tx)>=Nfits) return;

		for (ii=0;ii<NV*NV;ii++)M[ii]=0;
		for (ii=0;ii<NV*NV;ii++)Minv[ii]=0;

		//copy in data
		const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
		const float *s_varim = d_varim+(sz*sz*bx*BlockSize+sz*sz*tx);

		//initial values
		kernel_CenterofMass2D(sz, s_data, &newTheta[0], &newTheta[1]);
		kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &newTheta[3]);
		newTheta[2]=max(0.0, (Nmax-newTheta[3])*2*pi*PSFSigma*PSFSigma);
		newTheta[3] = max(newTheta[3],0.01);
		newTheta[4]=PSFSigma;

		maxJump[2]=max(newTheta[2],maxJump[2]);

		maxJump[3]=max(newTheta[3],maxJump[3]);

		for (ii=0;ii<NV;ii++)oldTheta[ii]=newTheta[ii];

		//updateFitValues
		newErr = 0;
		memset(jacobian,0,NV*sizeof(float));
		memset(hessian,0,NV*NV*sizeof(float));
		for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
			kernel_DerivativeGauss2D_sigma(ii,jj,newTheta,newDudt,&model);
			model +=s_varim[sz*jj+ii];
			data=s_data[sz*jj+ii]+s_varim[sz*jj+ii];	

			if (data>0)
				newErr = newErr + 2*((model-data)-data*log(model/data));
			else
			{
				newErr = newErr + 2*model;
				data = 0;
			}

			t1 = 1-data/model;
			for (l=0;l<NV;l++){
				jacobian[l]+=t1*newDudt[l];
			}

			t2 = data/pow(model,2);
			for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
				hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
				hessian[m*NV+l] = hessian[l*NV+m];
			}
		}

		for (kk=0;kk<iterations;kk++) {//main iterative loop

			if(abs((newErr-oldErr)/newErr)<TOLERANCE){
				//CONVERGED;
				break;
			}
			else{
				if(newErr>ACCEPTANCE*oldErr){
					//copy Fitdata

					for (i=0;i<NV;i++){
						newTheta[i]=oldTheta[i];
						newUpdate[i]=oldUpdate[i];
					}
					newLambda = oldLambda;
					newErr = oldErr;
					mu = max( (1 + newLambda*SCALE_UP)/(1 + newLambda),1.3f);         
					newLambda = SCALE_UP*newLambda;

				}
				else if(newErr<oldErr&&errFlag==0){
					newLambda = SCALE_DOWN*newLambda;
				    mu = 1+newLambda;
				}



				for (i=0;i<NV;i++){
					hessian[i*NV+i]=hessian[i*NV+i]*mu;
				}
				memset(L,0,NV*sizeof(float));
				memset(U,0,NV*sizeof(float));
				errFlag = kernel_cholesky(hessian,NV,L,U);
				if (errFlag ==0){
					for (i=0;i<NV;i++){
						oldTheta[i]=newTheta[i];
						oldUpdate[i] = newUpdate[i];
					}
					oldLambda = newLambda;
					oldErr=newErr;

					kernel_luEvaluate(L,U,jacobian,NV,newUpdate);	
					
					//updateFitParameters
					for (ll=0;ll<NV;ll++){
						if (newUpdate[ll]/oldUpdate[ll]< -0.5f){
							maxJump[ll] = maxJump[ll]*0.5;
						}
					    newUpdate[ll] = newUpdate[ll]/(1+fabs(newUpdate[ll]/maxJump[ll]));
						newTheta[ll] = newTheta[ll]-newUpdate[ll];
					}
					//restrict range
					newTheta[0] = max(newTheta[0],(float(sz)-1)/2-sz/4.0);
					newTheta[0] = min(newTheta[0],(float(sz)-1)/2+sz/4.0);
					newTheta[1] = max(newTheta[1],(float(sz)-1)/2-sz/4.0);
					newTheta[1] = min(newTheta[1],(float(sz)-1)/2+sz/4.0);
					newTheta[2] = max(newTheta[2],1.0);
					newTheta[3] = max(newTheta[3],0.01);
					newTheta[4] = max(newTheta[4],0.0);
					newTheta[4] = min(newTheta[4],sz/2.0f);


					newErr = 0;
					memset(jacobian,0,NV*sizeof(float));
					memset(hessian,0,NV*NV*sizeof(float));
					for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
						//calculating derivatives
						kernel_DerivativeGauss2D_sigma(ii,jj,newTheta,newDudt,&model);
						model +=s_varim[sz*jj+ii];
						data=s_data[sz*jj+ii]+s_varim[sz*jj+ii];			

						if (data>0)
							newErr = newErr + 2*((model-data)-data*log(model/data));
						else
						{
							newErr = newErr + 2*model;
							data = 0;
						}

						t1 = 1-data/model;
						for (l=0;l<NV;l++){
							jacobian[l]+=t1*newDudt[l];
						}

						t2 = data/pow(model,2);
						for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
							hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
							hessian[m*NV+l] = hessian[l*NV+m];
						}
					}
				}
				else
				{
					mu = max( (1 + newLambda*SCALE_UP)/(1 + newLambda),1.3f);         
					newLambda = SCALE_UP*newLambda;
				}
			}
		}
		//output iteration
		d_Parameters[Nfits*NV+BlockSize*bx+tx]=kk;

		// Calculating the CRLB and LogLikelihood
		Div=0.0f;
		for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
			
			kernel_DerivativeGauss2D_sigma(ii,jj,newTheta,newDudt,&model);
			model +=s_varim[sz*jj+ii];
			data=s_data[sz*jj+ii]+s_varim[sz*jj+ii];	

			//Building the Fisher Information Matrix
			for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
				M[kk*NV+ll]+= newDudt[ll]*newDudt[kk]/model;
				M[ll*NV+kk]=M[kk*NV+ll];
			}

			//LogLikelyhood
			if (model>0)
				if (data>0)Div+=data*log(model)-model-data*log(data)+data;
				else
					Div+=-model;
		}

		// Matrix inverse (CRLB=F^-1) and output assigments
		kernel_MatInvN(M, Minv, Diag, NV);
		//write to global arrays
		for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=newTheta[kk];
		for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
		d_LogLikelihood[BlockSize*bx+tx] = Div;

		return;
}

//*********************************************************************************************************************************************

__global__ void kernel_MLEFit_LM_z_sCMOS(const float *d_data, const float PSFSigma_x, const float Ax, const float Ay, const float Bx, 
	const float By, const float gamma, const float d, const float PSFSigma_y, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits, const float *d_varim){
			/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param PSFSigma_x sigma of the point spread function on the x axis
	 * \param Ax ???
	 * \param Ay ???
	 * \param Bx ???
	 * \param By ???
	 * \param gamma ???
	 * \param d ???
	 * \param PSFSigma_y sigma of the point spread function on the y axis
	 * \param sz nxn size of the subregion to fit
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 * \d_varim variance map of scmos
	 */

		const int NV=NV_PZ;
		float M[NV*NV],Diag[NV], Minv[NV*NV];
		int tx = threadIdx.x;
		int bx = blockIdx.x;
		int BlockSize = blockDim.x;
		int ii, jj, kk, ll, l, m, i;


		float model, data;
		float Div;
		float PSFy, PSFx;

		float newTheta[NV],oldTheta[NV];
		float newLambda = INIT_LAMBDA, oldLambda = INIT_LAMBDA, mu;
		float newUpdate[NV] = {1e13, 1e13, 1e13, 1e13, 1e13},oldUpdate[NV] = {1e13, 1e13, 1e13, 1e13, 1e13};
		float maxJump[NV]={1.0,1.0,100,20,2};
		float newDudt[NV] ={0};

		float newErr = 1e12, oldErr = 1e13;

		float jacobian[NV]={0};
		float hessian[NV*NV]={0};
		float t1,t2;

		float Nmax;
		int errFlag=0;
		float L[NV*NV] = {0}, U[NV*NV] = {0};


		//Prevent read/write past end of array
		if ((bx*BlockSize+tx)>=Nfits) return;

		for (ii=0;ii<NV*NV;ii++)M[ii]=0;
		for (ii=0;ii<NV*NV;ii++)Minv[ii]=0;

		//copy in data
		const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
		const float *s_varim = d_varim+(sz*sz*bx*BlockSize+sz*sz*tx);

		//initial values
		kernel_CenterofMass2D(sz, s_data, &newTheta[0], &newTheta[1]);
		kernel_GaussFMaxMin2D(sz, PSFSigma_x, s_data, &Nmax, &newTheta[3]);
		newTheta[2]=max(0.0, (Nmax-newTheta[3])*2*pi*PSFSigma_x*PSFSigma_y*sqrt(2.0f));
		newTheta[3] = max(newTheta[3],0.01);
		newTheta[4]=0;

		maxJump[2]=max(newTheta[2],maxJump[2]);

		maxJump[3]=max(newTheta[3],maxJump[3]);

		for (ii=0;ii<NV;ii++)oldTheta[ii]=newTheta[ii];

		//updateFitValues
		newErr = 0;
		memset(jacobian,0,NV*sizeof(float));
		memset(hessian,0,NV*NV*sizeof(float));
		for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
			 kernel_DerivativeIntGauss2Dz(ii, jj, newTheta, PSFSigma_x,PSFSigma_y, Ax,Ay,Bx,By, gamma, d, &PSFx, &PSFy, newDudt, NULL,&model);
			model +=s_varim[sz*jj+ii];
			data=s_data[sz*jj+ii]+s_varim[sz*jj+ii];	

			if (data>0)
				newErr = newErr + 2*((model-data)-data*log(model/data));
			else
			{
				newErr = newErr + 2*model;
				data = 0;
			}

			t1 = 1-data/model;
			for (l=0;l<NV;l++){
				jacobian[l]+=t1*newDudt[l];
			}

			t2 = data/pow(model,2);
			for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
				hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
				hessian[m*NV+l] = hessian[l*NV+m];
			}
		}

		for (kk=0;kk<iterations;kk++) {//main iterative loop

			if(abs((newErr-oldErr)/newErr)<TOLERANCE){
				//CONVERGED;
				break;
			}
			else{
				if(newErr>ACCEPTANCE*oldErr){
					//copy Fitdata

					for (i=0;i<NV;i++){
						newTheta[i]=oldTheta[i];
						newUpdate[i]=oldUpdate[i];
					}
					newLambda = oldLambda;
					newErr = oldErr;
					mu = max( (1 + newLambda*SCALE_UP)/(1 + newLambda),1.3f);         
					newLambda = SCALE_UP*newLambda;
				}
				else if(newErr<oldErr&&errFlag==0){
					newLambda = SCALE_DOWN*newLambda;
				    mu = 1+newLambda;
				}

				for (i=0;i<NV;i++){
					hessian[i*NV+i]=hessian[i*NV+i]*mu;
				}
				memset(L,0,NV*sizeof(float));
				memset(U,0,NV*sizeof(float));
				errFlag = kernel_cholesky(hessian,NV,L,U);
				if (errFlag ==0){
					for (i=0;i<NV;i++){
						oldTheta[i]=newTheta[i];
						oldUpdate[i] = newUpdate[i];
					}
					oldLambda = newLambda;
					oldErr=newErr;

					kernel_luEvaluate(L,U,jacobian,NV,newUpdate);	
					
					//updateFitParameters
					for (ll=0;ll<NV;ll++){
						if (newUpdate[ll]/oldUpdate[ll]< -0.5f){
							maxJump[ll] = maxJump[ll]*0.5;
						}
					    newUpdate[ll] = newUpdate[ll]/(1+fabs(newUpdate[ll]/maxJump[ll]));
						newTheta[ll] = newTheta[ll]-newUpdate[ll];
					}
					//restrict range
					newTheta[0] = max(newTheta[0],(float(sz)-1)/2-sz/4.0);
					newTheta[0] = min(newTheta[0],(float(sz)-1)/2+sz/4.0);
					newTheta[1] = max(newTheta[1],(float(sz)-1)/2-sz/4.0);
					newTheta[1] = min(newTheta[1],(float(sz)-1)/2+sz/4.0);
					newTheta[2] = max(newTheta[2],1.0);
					newTheta[3] = max(newTheta[3],0.01);


					newErr = 0;
					memset(jacobian,0,NV*sizeof(float));
					memset(hessian,0,NV*NV*sizeof(float));
					for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
						//calculating derivatives
						kernel_DerivativeIntGauss2Dz(ii, jj, newTheta, PSFSigma_x,PSFSigma_y, Ax,Ay,Bx,By, gamma, d, &PSFx, &PSFy, newDudt, NULL,&model);
						
						model +=s_varim[sz*jj+ii];
						data=s_data[sz*jj+ii]+s_varim[sz*jj+ii];			

						if (data>0)
							newErr = newErr + 2*((model-data)-data*log(model/data));
						else
						{
							newErr = newErr + 2*model;
							data = 0;
						}

						t1 = 1-data/model;
						for (l=0;l<NV;l++){
							jacobian[l]+=t1*newDudt[l];
						}

						t2 = data/pow(model,2);
						for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
							hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
							hessian[m*NV+l] = hessian[l*NV+m];
						}
					}
				}
				else
				{
					mu = max( (1 + newLambda*SCALE_UP)/(1 + newLambda),1.3f);         
					newLambda = SCALE_UP*newLambda;
				}
			}
		}
		d_Parameters[Nfits*NV+BlockSize*bx+tx]=kk;
		// Calculating the CRLB and LogLikelihood
		Div=0.0f;
		for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
		    kernel_DerivativeIntGauss2Dz(ii, jj, newTheta, PSFSigma_x,PSFSigma_y, Ax,Ay,Bx,By, gamma, d, &PSFx, &PSFy, newDudt, NULL,&model);
			model +=s_varim[sz*jj+ii];
			data=s_data[sz*jj+ii]+s_varim[sz*jj+ii];	

			//Building the Fisher Information Matrix
			for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
				M[kk*NV+ll]+= newDudt[ll]*newDudt[kk]/model;
				M[ll*NV+kk]=M[kk*NV+ll];
			}

			//LogLikelyhood
			if (model>0)
				if (data>0)Div+=data*log(model)-model-data*log(data)+data;
				else
					Div+=-model;
		}

		// Matrix inverse (CRLB=F^-1) and output assigments
		kernel_MatInvN(M, Minv, Diag, NV);
		//write to global arrays
		for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=newTheta[kk];
		for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
		d_LogLikelihood[BlockSize*bx+tx] = Div;

		return;
}

//*********************************************************************************************************************************************

__global__ void kernel_MLEFit_LM_sigmaxy_sCMOS(const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits, const float *d_varim){
			/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param PSFSigma sigma of the point spread function
	 * \param sz nxn size of the subregion to fit
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	  * \d_varim variance map of scmos
	 */

		const int NV=NV_PS2;
		float M[NV*NV],Diag[NV], Minv[NV*NV];
		int tx = threadIdx.x;
		int bx = blockIdx.x;
		int BlockSize = blockDim.x;
		int ii, jj, kk, ll, l, m, i;


		float model, data;
		float Div;

		float newTheta[NV],oldTheta[NV];
		float newLambda = INIT_LAMBDA, oldLambda = INIT_LAMBDA, mu;
		float newUpdate[NV] = {1e13, 1e13, 1e13, 1e13, 1e13, 1e13},oldUpdate[NV] = {1e13, 1e13, 1e13, 1e13, 1e13, 1e13};
		float maxJump[NV]={1.0,1.0,100,20,0.5,0.5};
		float newDudt[NV] ={0};

		float newErr = 1e12, oldErr = 1e13;

		float jacobian[NV]={0};
		float hessian[NV*NV]={0};
		float t1,t2;

		float Nmax;
		int errFlag=0;
		float L[NV*NV] = {0}, U[NV*NV] = {0};


		//Prevent read/write past end of array
		if ((bx*BlockSize+tx)>=Nfits) return;

		for (ii=0;ii<NV*NV;ii++)M[ii]=0;
		for (ii=0;ii<NV*NV;ii++)Minv[ii]=0;

		//copy in data
		const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
		const float *s_varim = d_varim+(sz*sz*bx*BlockSize+sz*sz*tx);

		//initial values
		kernel_CenterofMass2D(sz, s_data, &newTheta[0], &newTheta[1]);
		kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &newTheta[3]);
		newTheta[2]=max(0.0, (Nmax-newTheta[3])*2*pi*PSFSigma*PSFSigma);
		newTheta[3] = max(newTheta[3],0.01);
		newTheta[4]=PSFSigma;
		newTheta[5]=PSFSigma;

		maxJump[2]=max(newTheta[2],maxJump[2]);

		maxJump[3]=max(newTheta[3],maxJump[3]);

		for (ii=0;ii<NV;ii++)oldTheta[ii]=newTheta[ii];

		//updateFitValues
		newErr = 0;
		memset(jacobian,0,NV*sizeof(float));
		memset(hessian,0,NV*NV*sizeof(float));
		for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
			kernel_DerivativeGauss2D_sigmaxy( ii,  jj, newTheta, newDudt, &model);
			model +=s_varim[sz*jj+ii];
			data=s_data[sz*jj+ii]+s_varim[sz*jj+ii];	

			if (data>0)
				newErr = newErr + 2*((model-data)-data*log(model/data));
			else
			{
				newErr = newErr + 2*model;
				data = 0;
			}

			t1 = 1-data/model;
			for (l=0;l<NV;l++){
				jacobian[l]+=t1*newDudt[l];
			}

			t2 = data/pow(model,2);
			for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
				hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
				hessian[m*NV+l] = hessian[l*NV+m];
			}
		}

		for (kk=0;kk<iterations;kk++) {//main iterative loop

			if(abs((newErr-oldErr)/newErr)<TOLERANCE){
				//CONVERGED;
				break;
			}
			else{
				if(newErr>ACCEPTANCE*oldErr){
					//copy Fitdata

					for (i=0;i<NV;i++){
						newTheta[i]=oldTheta[i];
						newUpdate[i]=oldUpdate[i];
					}
					newLambda = oldLambda;
					newErr = oldErr;
					mu = max( (1 + newLambda*SCALE_UP)/(1 + newLambda),1.3f);         
					newLambda = SCALE_UP*newLambda;
				}
				else if(newErr<oldErr&&errFlag==0){
					newLambda = SCALE_DOWN*newLambda;
				    mu = 1+newLambda;
				}



				for (i=0;i<NV;i++){
					hessian[i*NV+i]=hessian[i*NV+i]*mu;
				}
				memset(L,0,NV*sizeof(float));
				memset(U,0,NV*sizeof(float));
				errFlag = kernel_cholesky(hessian,NV,L,U);
				if (errFlag ==0){
					for (i=0;i<NV;i++){
						oldTheta[i]=newTheta[i];
						oldUpdate[i] = newUpdate[i];
					}
					oldLambda = newLambda;
					oldErr=newErr;

					kernel_luEvaluate(L,U,jacobian,NV,newUpdate);	
					
					//updateFitParameters
					for (ll=0;ll<NV;ll++){
						if (newUpdate[ll]/oldUpdate[ll]< -0.5f){
							maxJump[ll] = maxJump[ll]*0.5;
						}
					    newUpdate[ll] = newUpdate[ll]/(1+fabs(newUpdate[ll]/maxJump[ll]));
						newTheta[ll] = newTheta[ll]-newUpdate[ll];
					}
					//restrict range
					newTheta[0] = max(newTheta[0],(float(sz)-1)/2-sz/4.0);
					newTheta[0] = min(newTheta[0],(float(sz)-1)/2+sz/4.0);
					newTheta[1] = max(newTheta[1],(float(sz)-1)/2-sz/4.0);
					newTheta[1] = min(newTheta[1],(float(sz)-1)/2+sz/4.0);
					newTheta[2] = max(newTheta[2],1.0);
					newTheta[3] = max(newTheta[3],0.01);
					newTheta[4] = max(newTheta[4],PSFSigma/10.0f);
					newTheta[5] = max(newTheta[5],PSFSigma/10.0f);


					newErr = 0;
					memset(jacobian,0,NV*sizeof(float));
					memset(hessian,0,NV*NV*sizeof(float));
					for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
						//calculating derivatives
						kernel_DerivativeGauss2D_sigmaxy( ii,  jj, newTheta, newDudt, &model);
						//kernel_DerivativeGauss2D_sigma(ii,jj,newTheta,newDudt,&model);
						model +=s_varim[sz*jj+ii];
						data=s_data[sz*jj+ii]+s_varim[sz*jj+ii];			

						if (data>0)
							newErr = newErr + 2*((model-data)-data*log(model/data));
						else
						{
							newErr = newErr + 2*model;
							data = 0;
						}

						t1 = 1-data/model;
						for (l=0;l<NV;l++){
							jacobian[l]+=t1*newDudt[l];
						}

						t2 = data/pow(model,2);
						for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
							hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
							hessian[m*NV+l] = hessian[l*NV+m];
						}
					}
				}
				else
				{
					mu = max( (1 + newLambda*SCALE_UP)/(1 + newLambda),1.3f);         
					newLambda = SCALE_UP*newLambda;
				}
			}
		}
		//output iteration
		d_Parameters[Nfits*NV+BlockSize*bx+tx]=kk;

		// Calculating the CRLB and LogLikelihood
		Div=0.0f;
		for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
			kernel_DerivativeGauss2D_sigmaxy( ii,  jj, newTheta, newDudt, &model);
			model +=s_varim[sz*jj+ii];
			data=s_data[sz*jj+ii]+s_varim[sz*jj+ii];	

			//Building the Fisher Information Matrix
			for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
				M[kk*NV+ll]+= newDudt[ll]*newDudt[kk]/model;
				M[ll*NV+kk]=M[kk*NV+ll];
			}

			//LogLikelyhood
			if (model>0)
				if (data>0)Div+=data*log(model)-model-data*log(data)+data;
				else
					Div+=-model;
		}

		// Matrix inverse (CRLB=F^-1) and output assigments
		kernel_MatInvN(M, Minv, Diag, NV);
		//write to global arrays
		for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=newTheta[kk];
		for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
		d_LogLikelihood[BlockSize*bx+tx] = Div;

		return;
}

//******************************************************************************************************

__global__ void kernel_splineMLEFit_z_sCMOS(const float *d_data,const float *d_coeff, const int spline_xsize, const int spline_ysize, const int spline_zsize, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,float initZ, const int Nfits, const float *d_varim){
			/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param d_coeff array of spline coefficients of the PSF model
	 * \param spline_xsize,spline_ysize,spline_zsize, x,y,z size of spline coefficients
	 * \param sz nxn size of the subregion to fit
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param initZ intial z position used for fitting
	 * \param Nfits number of subregions to fit
	 * \param d_varim variance map of sCMOS
	 */
	
   const int NV=NV_PSP;
    float M[NV*NV],Diag[NV], Minv[NV*NV];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll, l, m, i;
	int xstart, ystart, zstart;

	const float *s_coeff;
	s_coeff = d_coeff;

    float model, data;
    float Div;
    float newTheta[NV],oldTheta[NV];
	float newLambda = INIT_LAMBDA, oldLambda = INIT_LAMBDA, mu;
	float newUpdate[NV] = {1e13, 1e13, 1e13, 1e13, 1e13},oldUpdate[NV] = {1e13, 1e13, 1e13, 1e13, 1e13};
	float maxJump[NV]={1.0,1.0,100,20,2};
	float newDudt[NV] ={0};

	float newErr = 1e12, oldErr = 1e13;

	float off;
	float jacobian[NV]={0};
	float hessian[NV*NV]={0};
	float t1,t2;

	float Nmax;
	float xc,yc,zc;
	float delta_f[64]={0}, delta_dxf[64]={0}, delta_dyf[64]={0}, delta_dzf[64]={0};
	int errFlag=0;
	float L[NV*NV] = {0}, U[NV*NV] = {0};

    
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
    for (ii=0;ii<NV*NV;ii++)M[ii]=0;
    for (ii=0;ii<NV*NV;ii++)Minv[ii]=0;

    //copy in data
      const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
	  const float *s_varim = d_varim+(sz*sz*bx*BlockSize+sz*sz*tx);
    
    //initial values
   kernel_CenterofMass2D(sz, s_data, &newTheta[0], &newTheta[1]);
    kernel_GaussFMaxMin2D(sz, 1.5, s_data, &Nmax, &newTheta[3]);

	//central pixel of spline model
	newTheta[3] = max(newTheta[3],0.01);
	newTheta[2]= (Nmax-newTheta[3])/d_coeff[(int)(spline_zsize/2)*(spline_xsize*spline_ysize)+(int)(spline_ysize/2)*spline_xsize+(int)(spline_xsize/2)]*4;

    //newTheta[4]=float(spline_zsize)/2;
	newTheta[4]=initZ;

	maxJump[2]=max(newTheta[2],maxJump[2]);

	maxJump[3]=max(newTheta[3],maxJump[3]);

	maxJump[4]= max(spline_zsize/3.0f,maxJump[4]);

	for (ii=0;ii<NV;ii++)oldTheta[ii]=newTheta[ii];

	//updateFitValues
	xc = -1.0*((newTheta[0]-float(sz)/2)+0.5);
	yc = -1.0*((newTheta[1]-float(sz)/2)+0.5);

	off = floor((float(spline_xsize)+1.0-float(sz))/2);

	xstart = floor(xc);
	xc = xc-xstart;

	ystart = floor(yc);
	yc = yc-ystart;

	//zstart = floor(newTheta[4]);
	zstart = floor(newTheta[4]);
	zc = newTheta[4] -zstart;

	newErr = 0;
	memset(jacobian,0,NV*sizeof(float));
	memset(hessian,0,NV*NV*sizeof(float));
	kernel_computeDelta3D(xc, yc, zc, delta_f, delta_dxf, delta_dyf, delta_dzf);

	for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
		kernel_DerivativeSpline(ii+xstart+off,jj+ystart+off,zstart,spline_xsize,spline_ysize,spline_zsize,delta_f,delta_dxf,delta_dyf,delta_dzf,s_coeff,newTheta,newDudt,&model);
		model +=s_varim[sz*jj+ii];
		data=s_data[sz*jj+ii]+s_varim[sz*jj+ii];

		if (data>0)
			newErr = newErr + 2*((model-data)-data*log(model/data));
		else
		{
			newErr = newErr + 2*model;
			data = 0;
		}

		t1 = 1-data/model;
		for (l=0;l<NV;l++){
			jacobian[l]+=t1*newDudt[l];
		}

		t2 = data/pow(model,2);
		for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
			hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
			hessian[m*NV+l] = hessian[l*NV+m];
		}
	}

	for (kk=0;kk<iterations;kk++) {//main iterative loop

			if(abs((newErr-oldErr)/newErr)<TOLERANCE){
				//CONVERGED;
				break;
			}
			else{
				if(newErr>ACCEPTANCE*oldErr){
					//copy Fitdata
					
					for (i=0;i<NV;i++){
						newTheta[i]=oldTheta[i];
						newUpdate[i]=oldUpdate[i];
					}
					newLambda = oldLambda;
					newErr = oldErr;
					mu = max( (1 + newLambda*SCALE_UP)/(1 + newLambda),1.3f);         
					newLambda = SCALE_UP*newLambda;
				}
				else if(newErr<oldErr&&errFlag==0){
					newLambda = SCALE_DOWN*newLambda;
				    mu = 1+newLambda;
				}
				

				for (i=0;i<NV;i++){
					hessian[i*NV+i]=hessian[i*NV+i]*mu;
				}
				memset(L,0,NV*sizeof(float));
				memset(U,0,NV*sizeof(float));
				errFlag = kernel_cholesky(hessian,NV,L,U);
				if (errFlag ==0){
					for (i=0;i<NV;i++){
						oldTheta[i]=newTheta[i];
						oldUpdate[i] = newUpdate[i];
					}
					oldLambda = newLambda;
					oldErr=newErr;

					kernel_luEvaluate(L,U,jacobian,NV,newUpdate);	
					
					//updateFitParameters
					for (ll=0;ll<NV;ll++){
						if (newUpdate[ll]/oldUpdate[ll]< -0.5f){
							maxJump[ll] = maxJump[ll]*0.5;
						}
					    newUpdate[ll] = newUpdate[ll]/(1+fabs(newUpdate[ll]/maxJump[ll]));
						newTheta[ll] = newTheta[ll]-newUpdate[ll];
					}
					//restrict range
					newTheta[0] = max(newTheta[0],(float(sz)-1)/2-sz/4.0);
					newTheta[0] = min(newTheta[0],(float(sz)-1)/2+sz/4.0);
					newTheta[1] = max(newTheta[1],(float(sz)-1)/2-sz/4.0);
					newTheta[1] = min(newTheta[1],(float(sz)-1)/2+sz/4.0);
					newTheta[2] = max(newTheta[2],1.0);
					newTheta[3] = max(newTheta[3],0.01);
					newTheta[4] = max(newTheta[4],0.0);
					newTheta[4] = min(newTheta[4],float(spline_zsize));

					//updateFitValues
					xc = -1.0*((newTheta[0]-float(sz)/2)+0.5);
					yc = -1.0*((newTheta[1]-float(sz)/2)+0.5);

					xstart = floor(xc);
					xc = xc-xstart;

					ystart = floor(yc);
					yc = yc-ystart;

					zstart = floor(newTheta[4]);
					zc = newTheta[4] -zstart;


					newErr = 0;
					memset(jacobian,0,NV*sizeof(float));
					memset(hessian,0,NV*NV*sizeof(float));
					kernel_computeDelta3D(xc, yc, zc, delta_f, delta_dxf, delta_dyf, delta_dzf);
					for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
						//calculating derivatives
						kernel_DerivativeSpline(ii+xstart+off,jj+ystart+off,zstart,spline_xsize,spline_ysize,spline_zsize,delta_f,delta_dxf,delta_dyf,delta_dzf,s_coeff,newTheta,newDudt,&model);
						model +=s_varim[sz*jj+ii];
						data=s_data[sz*jj+ii]+s_varim[sz*jj+ii];

						if (data>0)
							newErr = newErr + 2*((model-data)-data*log(model/data));
						else
						{
							newErr = newErr + 2*model;
							data = 0;
						}

						t1 = 1-data/model;
						for (l=0;l<NV;l++){
							jacobian[l]+=t1*newDudt[l];
						}

						t2 = data/pow(model,2);
						for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
							hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
							hessian[m*NV+l] = hessian[l*NV+m];
						}
					}
				}
				else
				{
					mu = max( (1 + newLambda*SCALE_UP)/(1 + newLambda),1.3f);         
					newLambda = SCALE_UP*newLambda;
				}

			}


		
	}
	//output iteration time
    d_Parameters[Nfits*NV+BlockSize*bx+tx]=kk;
    
    // Calculating the CRLB and LogLikelihood
	Div=0.0;

	xc = -1.0*((newTheta[0]-float(sz)/2)+0.5);
	yc = -1.0*((newTheta[1]-float(sz)/2)+0.5);

	//off = (float(spline_xsize)+1.0-2*float(sz))/2;

	xstart = floor(xc);
	xc = xc-xstart;

	ystart = floor(yc);
	yc = yc-ystart;

	zstart = floor(newTheta[4]);
	zc = newTheta[4] -zstart;

	kernel_computeDelta3D(xc, yc, zc, delta_f, delta_dxf, delta_dyf, delta_dzf);

    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
		kernel_DerivativeSpline(ii+xstart+off,jj+ystart+off,zstart,spline_xsize,spline_ysize,spline_zsize,delta_f,delta_dxf,delta_dyf,delta_dzf,s_coeff,newTheta,newDudt,&model);
		model +=s_varim[sz*jj+ii];
		data=s_data[sz*jj+ii]+s_varim[sz*jj+ii];
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= newDudt[ll]*newDudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if (model>0)
            if (data>0)Div+=data*log(model)-model-data*log(data)+data;
            else
                Div+=-model;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
  
    
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=newTheta[kk];
   for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
   d_LogLikelihood[BlockSize*bx+tx] = Div;    
    
    return;
}
